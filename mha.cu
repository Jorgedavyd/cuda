#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <math.h>

/*
query: (batch_size, num_heads, sequence, head_dim)
values: (batch_size, num_heads, sequence, head_dim)
keys: (batch_size, num_heads, sequence, head_dim)

out = single_query@single_key.T/sqrt(d_model).apply(softmax)@v

threads(num_heads) -> blocks(num_batches) -> clusters (1)-> grid (1)
*/

namespace F = torch::nn::functional;

__global__ void kernel (
    torch::Tensor query,
    torch::Tensor key,
    torch::Tensor value,
    const float d_model,
    torch::Tensor& out
) {
    const unsigned int x = threadIdx.x + blockDim.x * blockIdx.x;
    const unsigned int batch = blockIdx.x;
    if (batch <= query.size(0)) {
        
    }
}

torch::Tensor forward (torch::Tensor& input, torch::Tensor& other) {
    const unsigned int batch_size = input.size(0);
    const unsigned int num_heads = input.size(1);
    torch::Tensor out = torch::empty_like(input);
    kernel<<<batch_size, num_heads>>>(input, other, out);
    return out;
}

int main (void) {
    torch::Tensor input = torch::randn({10, 10}, torch::device(torch::kCUDA));
    torch::Tensor other = torch::randn({10, 10}, torch::device(torch::kCUDA));
    
    auto out = forward(input, other);

    return 0;
}
